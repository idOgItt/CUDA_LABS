#include <hip/hip_runtime.h>

#include <cmath>
#include <cstdint>
#include <iostream>
#include <iterator>
#include <memory>
#include <stdexcept>
#include <type_traits>
#include <vector>

constexpr int32_t kThreadsPerBlock = 1024;

template <typename T>
struct CudaDeleter {
    void operator()(T* pointer) const noexcept {
        if (pointer) {
            hipFree(pointer);
        }
    }
};

template <typename T>
class CudaBuffer {
public:
    explicit CudaBuffer(const std::size_t kSize = 0) {
        if (kSize > 0) {
            T* temp_ptr = nullptr;
            if (hipMalloc(reinterpret_cast<void**>(&temp_ptr),
                           kSize * sizeof(T)) != hipSuccess) {
                throw std::runtime_error("Failed to allocate CUDA memory");
            }
            buffer_.reset(temp_ptr);
        }
    }

    [[nodiscard]] T* Get() const noexcept { return buffer_.get(); }

private:
    std::unique_ptr<T, CudaDeleter<T>> buffer_;
};

__global__ void VectorPerElemMinDouble(const double* first_vector,
                                       const double* second_vector,
                                       double* result,
                                       const int32_t* data_size) {
    extern __shared__ double shared_data[];

    double* shared_a = shared_data;
    double* shared_b = &shared_data[blockDim.x];

    int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t stride = gridDim.x * blockDim.x;

    for (; idx < *data_size; idx += stride) {
        shared_a[threadIdx.x] = first_vector[idx];
        shared_b[threadIdx.x] = second_vector[idx];

        __syncthreads();

        result[idx] = fmin(shared_a[threadIdx.x], shared_b[threadIdx.x]);

        __syncthreads();
    }
}

template <typename T,
          typename std::enable_if<std::is_arithmetic<T>::value, int>::type = 0>
class CudaVectorMin {
public:
    explicit CudaVectorMin(int32_t size)
        : vector_size_(size),
          device_vector_a_(size),
          device_vector_b_(size),
          device_result_(size),
          device_data_size_(size),
          graph_(nullptr),
          graphExec_(nullptr) {
        GetDeviceProperties();
        if (hipMemcpy(device_data_size_.Get(), &vector_size_, sizeof(int32_t),
                       hipMemcpyHostToDevice) != hipSuccess) {
            throw std::runtime_error("Failed to copy data size to device");
        }
        CreateCudaGraph();
    }

    [[nodiscard]] std::vector<T> Compute(const std::vector<T>& host_vector_a,
                                         const std::vector<T>& host_vector_b) {
        if (host_vector_a.size() != vector_size_ ||
            host_vector_b.size() != vector_size_) {
            throw std::invalid_argument("Vector sizes do not match");
        }

        if (hipMemcpy(device_vector_a_.Get(), host_vector_a.data(),
                       vector_size_ * sizeof(T),
                       hipMemcpyHostToDevice) != hipSuccess ||
            hipMemcpy(device_vector_b_.Get(), host_vector_b.data(),
                       vector_size_ * sizeof(T),
                       hipMemcpyHostToDevice) != hipSuccess) {
            throw std::runtime_error(
                "hipMemcpy failed while copying data to device");
        }

        RunCudaGraph();

        std::vector<T> host_result(vector_size_);
        if (hipMemcpy(host_result.data(), device_result_.Get(),
                       vector_size_ * sizeof(T),
                       hipMemcpyDeviceToHost) != hipSuccess) {
            throw std::runtime_error(
                "hipMemcpy failed while copying result back to host");
        }

        return host_result;
    }

    ~CudaVectorMin() {
        if (graphExec_) {
            hipGraphExecDestroy(graphExec_);
        }
        if (graph_) {
            hipGraphDestroy(graph_);
        }
    }

private:
    int32_t vector_size_;
    int32_t numSM_{};
    CudaBuffer<T> device_vector_a_;
    CudaBuffer<T> device_vector_b_;
    CudaBuffer<T> device_result_;
    CudaBuffer<int32_t> device_data_size_;
    hipGraph_t graph_{nullptr};
    hipGraphExec_t graphExec_{nullptr};

    void GetDeviceProperties() {
        hipDeviceProp_t device_prop;
        hipGetDeviceProperties(&device_prop, 0);
        numSM_ = device_prop.multiProcessorCount;
    }

    void CreateCudaGraph() {
        if (graphExec_) {
            hipGraphExecDestroy(graphExec_);
            graphExec_ = nullptr;
        }
        if (graph_) {
            hipGraphDestroy(graph_);
            graph_ = nullptr;
        }

        hipStream_t stream;
        hipStreamCreate(&stream);

        hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

        int32_t grid_size = numSM_ * 4;
        size_t shared_mem_size = 2 * kThreadsPerBlock * sizeof(double);
        VectorPerElemMinDouble<<<grid_size, kThreadsPerBlock, shared_mem_size,
                                 stream>>>(
            device_vector_a_.Get(), device_vector_b_.Get(),
            device_result_.Get(), device_data_size_.Get());

        hipStreamEndCapture(stream, &graph_);
        hipStreamDestroy(stream);

        hipError_t status =
            hipGraphInstantiate(&graphExec_, graph_, nullptr, nullptr, 0);
        if (status != hipSuccess) {
            throw std::runtime_error("hipGraphInstantiate failed: " +
                                     std::string(hipGetErrorString(status)));
        }
    }

    void RunCudaGraph() {
        if (!graphExec_) {
            throw std::runtime_error("Graph not instantiated!");
        }
        hipError_t status = hipGraphLaunch(graphExec_, 0);
        if (status != hipSuccess) {
            throw std::runtime_error("hipGraphLaunch failed: " +
                                     std::string(hipGetErrorString(status)));
        }
        status = hipDeviceSynchronize();
        if (status != hipSuccess) {
            throw std::runtime_error("hipDeviceSynchronize failed: " +
                                     std::string(hipGetErrorString(status)));
        }
    }
};

template <typename T,
          typename std::enable_if<std::is_arithmetic<T>::value, int>::type = 0>
std::vector<T> ReadVector(const int32_t kSize) {
    std::vector<T> vector(kSize);
    for (T& value : vector) {
        std::cin >> value;
    }
    return vector;
}

template <typename T,
          typename std::enable_if<std::is_arithmetic<T>::value, int>::type = 0>
void PrintVector(const std::vector<T>& vec) {
    constexpr int32_t kPrecision = 10;
    std::cout.precision(kPrecision);
    std::cout << std::scientific;

    std::copy(vec.begin(), vec.end() - 1,
              std::ostream_iterator<T>(std::cout, " "));
    std::cout << vec.back() << std::endl;
}

int32_t ReadInputSize() {
    int32_t input_size;
    std::cin >> input_size;

    constexpr int32_t kMaxSize = (1 << 25);
    if (input_size <= 0 || input_size >= kMaxSize) {
        throw std::invalid_argument("Incorrect size of input data");
    }

    return input_size;
}

int main() {
    try {
        const int32_t kInputSize = ReadInputSize();
        const std::vector<double> kHostVectorA = ReadVector<double>(kInputSize);
        const std::vector<double> kHostVectorB = ReadVector<double>(kInputSize);

        CudaVectorMin<double> cuda_vector_min(kInputSize);
        const std::vector<double> kResult =
            cuda_vector_min.Compute(kHostVectorA, kHostVectorB);

        PrintVector(kResult);
    } catch (const std::exception& exception) {
        std::cerr << "Error: " << exception.what() << std::endl;
        return 1;
    }

    return 0;
}