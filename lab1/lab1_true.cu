#include <hip/hip_runtime.h>

#include <cmath>
#include <cstdint>
#include <iostream>
#include <iterator>
#include <memory>
#include <stdexcept>
#include <type_traits>
#include <vector>

constexpr int32_t kGridSize = 32;
constexpr int32_t kThreadsPerBlock = 1024;

template <typename T>
struct CudaDeleter {
    void operator()(T* pointer) const noexcept {
        if (pointer) {
            hipFree(pointer);
        }
    }
};

template <typename T>
class CudaBuffer {
public:
    explicit CudaBuffer(const std::size_t kSize = 0) {
        if (kSize > 0) {
            T* temp_ptr = nullptr;
            if (hipMalloc(reinterpret_cast<void**>(&temp_ptr),
                           kSize * sizeof(T)) != hipSuccess) {
                throw std::runtime_error("Failed to allocate CUDA memory");
            }
            buffer_.reset(temp_ptr);
        }
    }

    [[nodiscard]] T* Get() const noexcept { return buffer_.get(); }

private:
    std::unique_ptr<T, CudaDeleter<T>> buffer_;
};

__global__ void VectorPerElemMinDouble(const double* first_vector,
                                       const double* second_vector,
                                       double* result, int32_t data_size) {
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    while (idx < data_size) {
        result[idx] = fmin(first_vector[idx], second_vector[idx]);
        idx += gridDim.x * blockDim.x;
    }
}

template <typename T,
          typename std::enable_if<std::is_arithmetic<T>::value, int>::type = 0>
class CudaVectorMin {
public:
    explicit CudaVectorMin(int32_t size)
        : vector_size_(size),
          device_vector_a_(size),
          device_vector_b_(size),
          device_result_(size) {}

    [[nodiscard]] std::vector<T> Compute(const std::vector<T>& host_vector_a,
                                         const std::vector<T>& host_vector_b) {
        if (host_vector_a.size() != vector_size_ ||
            host_vector_b.size() != vector_size_) {
            throw std::invalid_argument("Vector sizes do not match");
        }

        if (hipMemcpy(device_vector_a_.Get(), host_vector_a.data(),
                       vector_size_ * sizeof(T),
                       hipMemcpyHostToDevice) != hipSuccess ||
            hipMemcpy(device_vector_b_.Get(), host_vector_b.data(),
                       vector_size_ * sizeof(T),
                       hipMemcpyHostToDevice) != hipSuccess) {
            throw std::runtime_error(
                "hipMemcpy failed while copying data to device");
        }

        RunCudaKernel();

        std::vector<T> host_result(vector_size_);
        if (hipMemcpy(host_result.data(), device_result_.Get(),
                       vector_size_ * sizeof(T),
                       hipMemcpyDeviceToHost) != hipSuccess) {
            throw std::runtime_error(
                "hipMemcpy failed while copying result back to host");
        }

        return host_result;
    }

private:
    int32_t vector_size_;
    CudaBuffer<T> device_vector_a_;
    CudaBuffer<T> device_vector_b_;
    CudaBuffer<T> device_result_;

    void RunCudaKernel() {
        VectorPerElemMinDouble<<<kGridSize, kThreadsPerBlock>>>(
            device_vector_a_.Get(), device_vector_b_.Get(),
            device_result_.Get(), vector_size_);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            throw std::runtime_error(std::string("CUDA kernel launch error: ") +
                                     hipGetErrorString(err));
        }
    }
};

template <typename T,
          typename std::enable_if<std::is_arithmetic<T>::value, int>::type = 0>
std::vector<T> ReadVector(const int32_t kSize) {
    std::vector<T> vector(kSize);
    for (T& value : vector) {
        std::cin >> value;
    }
    return vector;
}

template <typename T,
          typename std::enable_if<std::is_arithmetic<T>::value, int>::type = 0>
void PrintVector(const std::vector<T>& vec) {
    constexpr int32_t kPrecision = 10;
    std::cout.precision(kPrecision);
    std::cout << std::scientific;

    std::copy(vec.begin(), vec.end() - 1,
              std::ostream_iterator<T>(std::cout, " "));
    std::cout << vec.back() << std::endl;
}

int32_t ReadInputSize() {
    int32_t input_size;
    std::cin >> input_size;

    constexpr int32_t kMaxSize = (1 << 25);
    if (input_size <= 0 || input_size >= kMaxSize) {
        throw std::invalid_argument("Incorrect size of input data");
    }

    return input_size;
}

int main() {
    try {
        const int32_t kInputSize = ReadInputSize();
        const std::vector<double> kHostVectorA = ReadVector<double>(kInputSize);
        const std::vector<double> kHostVectorB = ReadVector<double>(kInputSize);

        CudaVectorMin<double> cuda_vector_min(kInputSize);
        const std::vector<double> kResult =
            cuda_vector_min.Compute(kHostVectorA, kHostVectorB);

        PrintVector(kResult);
    } catch (const std::exception& exception) {
        std::cerr << "Error: " << exception.what() << std::endl;
        return 1;
    }

    return 0;
}
