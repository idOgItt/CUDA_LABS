#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include ""

#include <algorithm>
#include <cstdint>
#include <cstring>
#include <exception>
#include <fstream>
#include <iostream>
#include <memory>
#include <stdexcept>
#include <string>
#include <type_traits>
#include <vector>

// Подключаем OpenCV для работы с изображениями (например, PNG)
#include <opencv2/opencv.hpp>

struct IOPaths {
    std::string inputFile;
    std::string outputFile;
};

namespace cuda_utils {
    inline void CheckCudaError(hipError_t err, const char *msg) {
        if (err != hipSuccess) {
            throw std::runtime_error(std::string(msg) + ": " + hipGetErrorString(err));
        }
    }

    class CudaArray {
    public:
        CudaArray(std::size_t width, std::size_t height, const hipChannelFormatDesc &desc) {
            CheckCudaError(hipMallocArray(&array_, &desc, width, height), "Failed to allocate hipArray");
        }

        ~CudaArray() noexcept {
            if (array_) hipFreeArray(array_);
        }

        hipArray_t get() const noexcept { return array_; }

        void CopyFromHost(const void *hostData, std::size_t width, const std::size_t height,
                          const std::size_t pitch) const {
            CheckCudaError(hipMemcpy2DToArray(array_, 0, 0, hostData, pitch, pitch, height, hipMemcpyHostToDevice),
                           "Failed to copy image data to CUDA array");
        }

    private:
        hipArray_t array_{nullptr};
    };

    class CudaTexture {
    public:
        CudaTexture(const hipResourceDesc &res, const hipTextureDesc &tex) {
            CheckCudaError(hipCreateTextureObject(&texObj_, &res, &tex, NULL), "Failed to create texture object");
        }

        ~CudaTexture() noexcept {
            if (texObj_) hipDestroyTextureObject(texObj_);
        }

        hipTextureObject_t get() const noexcept { return texObj_; }

    private:
        hipTextureObject_t texObj_{0};
    };

    template<typename T>
    class CudaBuffer {
    public:
        explicit CudaBuffer(const std::size_t count) {
            T *temp = nullptr;
            if (hipMalloc(&temp, count * sizeof(T)) != hipSuccess) {
                throw std::runtime_error("Failed to allocate CUDA memory");
            }
            buffer_.reset(temp);
        }

        T *Get() const noexcept { return buffer_.get(); }

        void CopyFromHost(const std::vector<T> &hostVector) {
            CopyToDevice(hostVector.data(), hostVector.size());
        }

        void CopyToHost(std::vector<T> &hostVector) {
            CopyFromDevice(hostVector.data(), hostVector.size());
        }

    private:
        struct CudaDeleter {
            void operator()(T *ptr) const noexcept {
                if (ptr) hipFree(ptr);
            }
        };

        std::unique_ptr<T, CudaDeleter> buffer_;

        void CopyToDevice(const T *hostPtr, std::size_t size) {
            CheckCudaError(hipMemcpy(buffer_.get(), hostPtr, size * sizeof(T), hipMemcpyHostToDevice),
                           "Failed to copy to device");
        }

        void CopyFromDevice(T *hostPtr, std::size_t size) {
            CheckCudaError(hipMemcpy(hostPtr, buffer_.get(), size * sizeof(T), hipMemcpyDeviceToHost),
                           "Failed to copy from device");
        }
    };

    class CudaGraph {
    public:
        CudaGraph() {
            CheckCudaError(hipStreamCreate(&stream_), "Failed to create stream");
        }

        ~CudaGraph() noexcept {
            if (graphExec_) hipGraphExecDestroy(graphExec_);
            if (graph_) hipGraphDestroy(graph_);
            hipStreamDestroy(stream_);
        }

        template<typename Func>
        void Capture(Func launcher) {
            CheckCudaError(hipStreamBeginCapture(stream_, hipStreamCaptureModeGlobal),
                           "Failed to begin graph capture");
            launcher(stream_);
            CheckCudaError(hipStreamEndCapture(stream_, &graph_), "Failed to end graph capture");
            CheckCudaError(hipGraphInstantiate(&graphExec_, graph_, NULL, NULL, 0), "Failed to instantiate graph");
        }

        void Launch() {
            CheckCudaError(hipGraphLaunch(graphExec_, stream_), "Failed to launch graph");
            CheckCudaError(hipStreamSynchronize(stream_), "Failed to synchronize stream");
        }

        hipStream_t GetStream() const noexcept { return stream_; }

    private:
        hipStream_t stream_{nullptr};
        hipGraph_t graph_{nullptr};
        hipGraphExec_t graphExec_{nullptr};
    };

    class CudaEvent {
    public:
        CudaEvent() {
            CheckCudaError(hipEventCreate(&event_), "Failed to create CUDA event");
        }

        ~CudaEvent() noexcept {
            if (event_) hipEventDestroy(event_);
        }

        hipEvent_t get() const noexcept { return event_; }

    private:
        hipEvent_t event_{nullptr};
    };
} // namespace cuda_utils

__global__ void sobelFilterKernel(uchar4 *outputImg, const std::size_t imgW, const std::size_t imgH,
                                  hipTextureObject_t texObj) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    constexpr float kernelX[3][3] = {{-1.f, 0.f, 1.f}, {-2.f, 0.f, 2.f}, {-1.f, 0.f, 1.f}};
    constexpr float kernelY[3][3] = {{1.f, 2.f, 1.f}, {0.f, 0.f, 0.f}, {-1.f, -2.f, -1.f}};

    const int stepX = blockDim.x * gridDim.x;
    const int stepY = blockDim.y * gridDim.y;

    for (; row < imgH; row += stepY) {
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        for (; col < imgW; col += stepX) {
            float gradX = 0.f, gradY = 0.f;
            for (int dy = -1; dy <= 1; ++dy) {
                for (int dx = -1; dx <= 1; ++dx) {
                    uchar4 pixel = tex2D<uchar4>(texObj, col + dx, row + dy);
                    float intensity = 0.299f * pixel.x + 0.587f * pixel.y + 0.114f * pixel.z;
                    gradX += intensity * kernelX[dy + 1][dx + 1];
                    gradY += intensity * kernelY[dy + 1][dx + 1];
                }
            }
            float edgeValue = sqrtf(gradX * gradX + gradY * gradY);
            edgeValue = fminf(fmaxf(edgeValue, 0.f), 255.0001f);
            uchar4 centerPixel = tex2D<uchar4>(texObj, col, row);
            outputImg[row * imgW + col] = {
                static_cast<unsigned char>(edgeValue),
                static_cast<unsigned char>(edgeValue),
                static_cast<unsigned char>(edgeValue),
                centerPixel.w
            };
        }
    }
}

class SobelExecutor {
public:
    SobelExecutor(const uint32_t width, const uint32_t height, const std::vector<uchar4> &input)
        : width_(width), height_(height),
          numPixels_(static_cast<std::size_t>(width) * height),
          deviceOutput_(numPixels_),
          textureArray_(width, height, hipCreateChannelDesc<uchar4>()),
          textureObject_(CreateTextureObject(textureArray_.get())) {
        textureArray_.CopyFromHost(input.data(), width, height, width * sizeof(uchar4));
        InitGraph();
    }

    std::vector<uchar4> Run() {
#ifdef ENABLE_TIMING
        const cuda_utils::CudaEvent start;
        const cuda_utils::CudaEvent stop;
        cuda_utils::CheckCudaError(hipEventRecord(start.get(), graph_.GetStream()), "Failed to record start event");
#endif

        graph_.Launch();

#ifdef ENABLE_TIMING
        cuda_utils::CheckCudaError(hipEventRecord(stop.get(), graph_.GetStream()), "Failed to record stop event");
        cuda_utils::CheckCudaError(hipEventSynchronize(stop.get()), "Failed to sync stop event");

        [[maybe_unused]] float elapsedMs = 0.f;
        cuda_utils::CheckCudaError(hipEventElapsedTime(&elapsedMs, start.get(), stop.get()), "Failed to measure time");
        std::cerr << "Execution time (ms): " << elapsedMs << std::endl;
#endif

        std::vector<uchar4> result(numPixels_);
        deviceOutput_.CopyToHost(result);
        return result;
    }

private:
    void InitGraph() {
        constexpr dim3 kThreads{32, 32};
        constexpr uint32_t kMaxGrid = 32;

        dim3 blocks((width_ + kThreads.x - 1) / kThreads.x,
                    (height_ + kThreads.y - 1) / kThreads.y);

        blocks.x = std::min(blocks.x, kMaxGrid);
        blocks.y = std::min(blocks.y, kMaxGrid);

        graph_.Capture([&](hipStream_t stream) {
            sobelFilterKernel<<<blocks, kThreads, 0, stream>>>(deviceOutput_.Get(), width_, height_,
                                                               textureObject_.get());
        });
    }

    static cuda_utils::CudaTexture CreateTextureObject(const hipArray_t array) {
        hipResourceDesc resDesc{};
        resDesc.resType = hipResourceTypeArray;
        resDesc.res.array.array = array;

        hipTextureDesc texDesc{};
        texDesc.addressMode[0] = hipAddressModeClamp;
        texDesc.addressMode[1] = hipAddressModeClamp;
        texDesc.filterMode = hipFilterModePoint;
        texDesc.readMode = hipReadModeElementType;
        texDesc.normalizedCoords = 0;

        return cuda_utils::CudaTexture(resDesc, texDesc);
    }

    uint32_t width_, height_;
    std::size_t numPixels_;
    cuda_utils::CudaBuffer<uchar4> deviceOutput_;
    cuda_utils::CudaArray textureArray_;
    cuda_utils::CudaTexture textureObject_;
    cuda_utils::CudaGraph graph_;
};

//
// Функция ReadImage использует OpenCV для загрузки изображений (например, PNG)
// и преобразует их в формат RGBA (uchar4), необходимый для обработки на GPU.
//
void ReadImage(const std::string &path, uint32_t &width, uint32_t &height, std::vector<uchar4> &data) {
    cv::Mat img = cv::imread(path, cv::IMREAD_UNCHANGED);
    if (img.empty()) {
        throw std::runtime_error("Failed to load image: " + path);
    }

    cv::Mat imgRGBA;
    if (img.channels() == 1) {
        cv::cvtColor(img, imgRGBA, cv::COLOR_GRAY2RGBA);
    } else if (img.channels() == 3) {
        // OpenCV по умолчанию загружает изображение в формате BGR
        cv::cvtColor(img, imgRGBA, cv::COLOR_BGR2RGBA);
    } else if (img.channels() == 4) {
        // Преобразуем BGRA в RGBA
        cv::cvtColor(img, imgRGBA, cv::COLOR_BGRA2RGBA);
    } else {
        throw std::runtime_error("Unsupported image format");
    }

    if (!imgRGBA.isContinuous()) {
        imgRGBA = imgRGBA.clone();
    }

    width = static_cast<uint32_t>(imgRGBA.cols);
    height = static_cast<uint32_t>(imgRGBA.rows);
    const std::size_t numPixels = static_cast<std::size_t>(width) * height;
    data.resize(numPixels);
    std::memcpy(data.data(), imgRGBA.data, numPixels * sizeof(uchar4));
}

//
// Функция WriteImage сохраняет обработанное изображение в формате PNG.
// Если указанное имя файла не содержит расширения ".png", оно дополняется автоматически.
//
void WriteImage(const std::string &path, const uint32_t width, const uint32_t height, const std::vector<uchar4> &data) {
    std::string outPath = path;
    // Проверяем, содержит ли имя файла расширение ".png"
    if (outPath.size() < 4 || outPath.substr(outPath.size() - 4) != ".png") {
        outPath += ".png";
    }

    // Создаем Mat из обработанных данных (RGBA)
    cv::Mat imgRGBA(height, width, CV_8UC4, const_cast<uchar4*>(data.data()));
    cv::Mat imgBGRA;
    cv::cvtColor(imgRGBA, imgBGRA, cv::COLOR_RGBA2BGRA);
    if (!cv::imwrite(outPath, imgBGRA)) {
        throw std::runtime_error("Failed to write image: " + outPath);
    }
}

IOPaths ReadInputOutputPaths() {
    IOPaths paths;
    std::cin >> paths.inputFile >> paths.outputFile;
    return paths;
}

int main() {
    try {
        IOPaths paths = ReadInputOutputPaths();
        const std::string &inputFile = paths.inputFile;
        const std::string &outputFile = paths.outputFile;

        uint32_t width = 0, height = 0;
        std::vector<uchar4> imageData;
        ReadImage(inputFile, width, height, imageData);

        std::cerr << "Image size: " << width << " x " << height << std::endl;

        SobelExecutor executor(width, height, imageData);
        std::vector<uchar4> result = executor.Run();

        WriteImage(outputFile, width, height, result);
        return 0;
    } catch (const std::exception &ex) {
        std::cerr << "Error: " << ex.what() << std::endl;
        return 1;
    }
}
